
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

__global__ void add(int *a, int *b, int *c, int num) {
	if ( threadIdx.x < num ) 
		c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main(int argc, char ** argv) {
	
	int num = 10;
	int a[num], b[num], c[num];
	
	for (int i=0;i<num;++i) a[i] = i;
	for (int i=0;i<num;++i) b[i] = i * i;

	int *agpu, *bgpu, *cgpu;
	
	hipMalloc((void**)&agpu, num * sizeof(int));
	hipMalloc((void**)&bgpu, num * sizeof(int));
	hipMalloc((void**)&cgpu, num * sizeof(int));

	hipMemcpy(agpu, a, num * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(bgpu, b, num * sizeof(int), hipMemcpyHostToDevice);

	// 加法
	add<<<1, 10>>>(agpu, bgpu, cgpu, num);
	hipMemcpy(c, cgpu, num * sizeof(int), hipMemcpyDeviceToHost);
	
	printf("add:\n");
	for (int i=0;i<num;++i) printf("%d + %d = %d\n", a[i], b[i], c[i]);

	hipFree(agpu); 
	hipFree(bgpu); 
	hipFree(cgpu); 
    hipDeviceReset(); 
}