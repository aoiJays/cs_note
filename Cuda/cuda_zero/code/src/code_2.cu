
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\"\n", \
                __FILE__, __LINE__, err, hipGetErrorString(err), #call); \
        exit(EXIT_FAILURE); \
    } \
} while (0)


__global__ void conv(float * img, float * kernel, float * res, int w, int h, int kernel_size) {
	
	// 边缘检测
	if ( threadIdx.x + blockIdx.x * blockDim.x >= w * h ) return;

	int x = (threadIdx.x + blockIdx.x * blockDim.x) / w;
	int y = (threadIdx.x + blockIdx.x * blockDim.x) % w;

	res[x * w + y] = 0;
	
	for (int i=0;i<kernel_size;++i)
		for (int j=0;j<kernel_size;++j) {
			int curx = x - kernel_size/2 + i;
			int cury = y - kernel_size/2 + j;
			if ( curx < 0 || curx >= h || cury < 0 || cury >= w ) continue;
			res[x * w + y] += kernel[i * kernel_size + j] * img[curx * w + cury]; 
		}	
}

int main() {

	// 定义图像
	int width = 1920, height = 1080;
	float *img = new float[width * height];
	float *res = new float[width * height];
	
	for (int i=0;i<height;++i)
		for (int j=0;j<width;++j)
			img[i * width + j] = (i + j) % 256;
	
	// 定义卷积核
	int kernel_size = 3;
	float *kernel = new float[kernel_size * kernel_size];
	for (int i=0;i<kernel_size;++i)
		for (int j=0;j<kernel_size;++j)
			kernel[i * kernel_size + j] = j - 1;
	
	// debug查看前10*10的矩阵
	puts("Img");
	for (int i=0;i<10;++i) {
		for (int j=0;j<10;++j)
			printf("%2.0f ",  img[i * width + j]);
		puts("");
	}

	puts("Kernel");
	for (int i=0;i<kernel_size;++i) {
		for (int j=0;j<kernel_size;++j)
			printf("%2.0f ",  kernel[i * kernel_size + j]);
		puts("");
	}

	// 申请显存
	float * img_gpu, *kernel_gpu, *res_gpu;
	CUDA_CHECK(hipMalloc((void**)&img_gpu, width*height*sizeof(float)));
	CUDA_CHECK(hipMalloc((void**)&kernel_gpu, kernel_size*kernel_size*sizeof(float)));
	CUDA_CHECK(hipMalloc((void**)&res_gpu, width*height*sizeof(float)));

	// 内存->显存
	CUDA_CHECK(hipMemcpy(img_gpu, img, width*height*sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(kernel_gpu, kernel, kernel_size*kernel_size*sizeof(float), hipMemcpyHostToDevice));

	// 获取可定义的block和thread数量情况
	// 方便定义并行数量
	auto getMaxThreadCount = [&](int deviceID) {
		hipDeviceProp_t prop;
		CUDA_CHECK(hipGetDeviceProperties(&prop, deviceID));
		printf("maxGridSize: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("maxThreadsPerBlock: %d\n", prop.maxThreadsPerBlock);
		return prop.maxThreadsPerBlock;
	};

	// maxGridSize: (2147483647, 65535, 65535)
	// maxThreadsPerBlock: 1024
	int threadNum = getMaxThreadCount(0);
	
	// 把1920*1080个像素分成若干块，每块1024大小（Thread数量）
	int blockNum = ( width * height + threadNum - 1 ) / threadNum; // 向上取整
	conv<<<blockNum, threadNum>>>(img_gpu, kernel_gpu, res_gpu, width, height, kernel_size);

	// 显存->内存
	CUDA_CHECK(hipMemcpy(res, res_gpu, width*height*sizeof(float), hipMemcpyDeviceToHost));
	
	// debug查看前10*10的矩阵
	puts("Res");
	for (int i=0;i<10;++i) {
		for (int j=0;j<10;++j)
			printf("%2.0f ",  res[i * width + j]);
		puts("");
	}	


	CUDA_CHECK(hipFree(img_gpu));
	CUDA_CHECK(hipFree(kernel_gpu));
	CUDA_CHECK(hipFree(res_gpu));
	delete[] img; delete[] kernel; delete[] res;
    return 0;
}
